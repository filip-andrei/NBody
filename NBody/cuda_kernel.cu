#include "hip/hip_runtime.h"
#include "cuda_kernel.cuh"
#include <cmath>
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>
#include <hiprand.h>
#include <>



__device__ const float kmPerPc = 3.0857e13;	//	Kilometers per Parsec
__device__ const float G = 4.302e-3;		//	Gravitational constant in ( pc / SM ) * (km/s)^2

//	Modified bessel functions I0,I1,K0,K1
__device__ float mbessi0(float x) {
   float ax,ans;
   float y;

   if ((ax=fabs(x)) < 3.75f) {
      y=x/3.75f,y=y*y;
      ans=1.0f+y*(3.5156229f+y*(3.0899424f+y*(1.2067492f
         +y*(0.2659732f+y*(0.360768e-1f+y*0.45813e-2f)))));
   } else {
      y=3.75f/ax;
      ans=(exp(ax)/sqrt(ax))*(0.39894228f+y*(0.1328592e-1f
         +y*(0.225319e-2f+y*(-0.157565e-2f+y*(0.916281e-2f
         +y*(-0.2057706e-1f+y*(0.2635537e-1f+y*(-0.1647633e-1f
         +y*0.392377e-2f))))))));
   }
   return ans;
}

__device__ float mbessi1(float x) {
   float ax,ans;
   float y;


   if ((ax=fabs(x)) < 3.75) {
      y=x/3.75,y=y*y;
      ans=ax*(0.5+y*(0.87890594+y*(0.51498869+y*(0.15084934
         +y*(0.2658733e-1+y*(0.301532e-2+y*0.32411e-3))))));
   } else {
      y=3.75/ax;
      ans=0.2282967e-1+y*(-0.2895312e-1+y*(0.1787654e-1
         -y*0.420059e-2));
      ans=0.39894228+y*(-0.3988024e-1+y*(-0.362018e-2
         +y*(0.163801e-2+y*(-0.1031555e-1+y*ans))));
      ans *= (exp(ax)/sqrt(ax));
   }
   return x < 0.0 ? -ans : ans;
}

__device__ float mbessk0(float x) {
   float y,ans;

   if (x <= 2.0) {
      y=x*x/4.0;
      ans=(-log(x/2.0)*mbessi0(x))+(-0.57721566+y*(0.42278420
         +y*(0.23069756+y*(0.3488590e-1+y*(0.262698e-2
         +y*(0.10750e-3+y*0.74e-5))))));
   } else {
      y=2.0/x;
      ans=(exp(-x)/sqrt(x))*(1.25331414+y*(-0.7832358e-1
         +y*(0.2189568e-1+y*(-0.1062446e-1+y*(0.587872e-2
         +y*(-0.251540e-2+y*0.53208e-3))))));
   }
   return ans;
}

__device__ float mbessk1(float x) {
   float y,ans;

   if (x <= 2.0) {
      y=x*x/4.0;
      ans=(log(x/2.0)*mbessi1(x))+(1.0/x)*(1.0+y*(0.15443144
         +y*(-0.67278579+y*(-0.18156897+y*(-0.1919402e-1
         +y*(-0.110404e-2+y*(-0.4686e-4)))))));
   } else {
      y=2.0/x;
      ans=(exp(-x)/sqrt(x))*(1.25331414+y*(0.23498619
         +y*(-0.3655620e-1+y*(0.1504268e-1+y*(-0.780353e-2
         +y*(0.325614e-2+y*(-0.68245e-3)))))));
   }
   return ans;
}

//	Get mass of dark matter contained in radius r
//	according to Hernquist density profile
__device__ float dmMassAtRadius(float r, 
								float Mdm,	//	Total dark matter mass in galaxy
								float a)	//	Scale radius for Hernquist density profile
{
	return (Mdm * r * r) / pow(r + a, 2);
}

__global__ void cudaGenBodies(float *d_pos, float *d_vel, float *d_rands, int NUM_PARTICLES, float Ms, float Rs, float Mdm, float Rdm){

	int threadId = threadIdx.x;
	int blockId = blockIdx.x;

	int globalId = blockId * blockDim.x + threadId;

	if(globalId < NUM_PARTICLES){
		int baseIndex = globalId * 3;

		float x = d_rands[baseIndex];
		float y = d_rands[baseIndex+1];
		float z = d_rands[baseIndex+2];

		//	Set position

		float rx = -Rs * log(1.0f - x);

		float Sz = -(1.0f/2.0f) * (0.1f * Rs) * log(-((z-1)/z));		
		float Sx = sqrt(rx*rx) * cos(2.0f * 3.1416f * y);
		float Sy = sqrt(rx*rx) * sin(2.0f * 3.1416f * y);

		d_pos[baseIndex] = Sx;
		d_pos[baseIndex+1] = Sy;
		d_pos[baseIndex+2] = Sz;

		//	Set velocity

		float realRad = sqrt(Sx * Sx + Sy * Sy + Sz * Sz);
		float t = realRad / (2.0f * Rs);
		float absVel = sqrt( (G * dmMassAtRadius(realRad, Mdm, Rdm)) / realRad + ((2.0f * G * Ms) / Rs) * t * t * (mbessi0(t)*mbessk0(t) - mbessi1(t)*mbessk1(t)) );
		
		float3 velUnitVector = make_float3( - Sy / sqrt(Sx*Sx+Sy*Sy) , Sx / sqrt(Sx*Sx+Sy*Sy), 0);

		float3 velVector = make_float3(velUnitVector.x * absVel, velUnitVector.y * absVel, velUnitVector.z * absVel);

		d_vel[baseIndex] = velVector.x;
		d_vel[baseIndex+1] = velVector.y;
		d_vel[baseIndex+2] = velVector.z;
	}	
}

void genBodies(GLuint posVBO, GLuint velVBO, int NUM_PARTICLES, float Ms, float Rs, float Mdm, float Rdm){

	cudaGLRegisterBufferObject(posVBO);
	cudaGLRegisterBufferObject(velVBO);
	float *d_pos;
	float *d_vel;
	cudaGLMapBufferObject( (void **)&d_pos, posVBO);
	cudaGLMapBufferObject( (void **)&d_vel, velVBO);


	int blockSize = 256;
	int blocks = NUM_PARTICLES / blockSize + (NUM_PARTICLES % blockSize == 0 ? 0:1);

	hiprandGenerator_t gen;
	hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
	hiprandSetPseudoRandomGeneratorSeed(gen, 1234ULL);

	float *d_randoms;
	hipMalloc(&d_randoms, sizeof(float) * 3 * NUM_PARTICLES);

	hiprandGenerateUniform(gen, d_randoms, NUM_PARTICLES * 3);

	cudaGenBodies<<<blocks, blockSize>>>(d_pos, d_vel, d_randoms, NUM_PARTICLES, Ms, Rs, Mdm, Rdm);

	hipFree(d_randoms);
	hiprandDestroyGenerator(gen);

	cudaGLUnmapBufferObject(posVBO);
	cudaGLUnmapBufferObject(velVBO);
}